#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


// Helper function for checking CUDA errors
#define CUDA_CHECK(call) \
do { \
    hipError_t result = call; \
    if (result != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", \
        __FILE__, __LINE__, result, hipGetErrorString(result), #call); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

__global__ void calc_pairwise_dist_kernel(
    double* grid_pos, double* coords, int N_coords, int N_grid_points, 
    double* dists
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
  
    if (i < N_grid_points && j < N_coords) {
        double dx, dy, dz;

        dx = grid_pos[i * 3] - coords[j * 3];
        dy = grid_pos[i * 3 + 1] - coords[j * 3 + 1];
        dz = grid_pos[i * 3 + 2] - coords[j * 3 + 2];

        dists[i * N_coords + j] = sqrtf(dx * dx + dy * dy + dz * dz);
    }
}

__device__ double calc_point_elec_potential(
    double dist, double elec_const, double charge, double rc, double alpha,
    double elec_rep_max, double elec_attr_max
) {
    double cur_potential;
    
    if (dist > rc) {
        cur_potential = elec_const / (dist * dist);
    } else {
        double alpha_tmp = alpha * dist * dist;
        if (charge > 0) {
            cur_potential = elec_rep_max - alpha_tmp;
        } else {
            cur_potential = elec_attr_max + alpha_tmp;
        }
    }
    
    return cur_potential;
}

__global__ void gen_elec_grid_kernel(
    const double* dists, const double* charges, const double cc_elec,
    const double rad_dielec_const, const double elec_rep_max, 
    const double elec_attr_max,
    const int N_coords, const int N_grid_points, double* electrostat_grid
) { 
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N_grid_points) {
        double elec_const, rc, alpha;
        double emax_tmp;
        double cur_grid_val = 0.0;
        for (int j = 0; j < N_coords; j++) {
            double dist = dists[i * N_coords + j];

            elec_const = cc_elec * charges[j] / rad_dielec_const;

            if (charges[j] > 0) {
                emax_tmp = elec_rep_max;
            } else {
                emax_tmp = elec_attr_max;
            }
            rc = sqrt(2.0 * fabs(elec_const / emax_tmp));
            alpha = fabs(emax_tmp / (2.0 * rc * rc));

            double cur_potential = calc_point_elec_potential(
                dist, elec_const, charges[j], rc, alpha, 
                elec_rep_max, elec_attr_max
            );
            cur_grid_val += cur_potential;
        }
        electrostat_grid[i] = cur_grid_val;
    }
}

__device__ double calc_point_vdw_potential(
    double dist, double eps_sqrt, double r_min, double probe_radius, 
    double vwd_softcore_max, double rc_vdw, double beta
) {
    double cur_potential;

    double r_min_over_dist = r_min / dist;
    if (dist > rc_vdw) {
        cur_potential = (
            eps_sqrt * (
                powf(r_min_over_dist, 12.0) - 2.0 * powf(r_min_over_dist, 6.0)
            )
        );
    } else {
        cur_potential = (
            vwd_softcore_max * (1.0 - 0.5 * powf((dist / rc_vdw), beta))
        );
    }

    return cur_potential;
}

__global__ void gen_vdw_grid_kernel(
    double* dists, double* epsilons, double* vdw_rs, double probe_radius, 
    double vwd_softcore_max, int N_coords, int N_grid_points, double* vdw_grid
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < N_grid_points) {
        double r_min, eps_sqrt, vdwconst, rc_vdw, beta;
        double cur_grid_val = 0.0;
        for (int j = 0; j < N_coords; j++) {
            double dist = dists[i * N_coords + j];

            r_min = vdw_rs[j] + probe_radius;
            eps_sqrt = sqrt(fabs(epsilons[j]));
            vdwconst = 1.0 + sqrt(1.0 + 0.5 * fabs(vwd_softcore_max) / eps_sqrt);
            rc_vdw = r_min * powf(vdwconst, -1.0 / 6.0);
            beta = 24.0 * eps_sqrt / 
            vwd_softcore_max * (vdwconst * vdwconst - vdwconst);

            double cur_potential = calc_point_vdw_potential(
                dist, eps_sqrt, r_min, probe_radius, 
                vwd_softcore_max, rc_vdw, beta
            );
            cur_grid_val += cur_potential;
        }
        vdw_grid[i] = cur_grid_val;
    }
}

extern "C"
void calc_pairwise_dist(
    double* host_grid_pos, double* host_coords, 
    const int N_coords, const int N_grid_points, double* host_dists
) {
    // Allocate memory on the device
    double* device_grid_pos;
    double* device_coords;
    double* device_dists;

    hipMalloc((void**)&device_grid_pos, N_grid_points * 3 * sizeof(double));
    hipMalloc((void**)&device_coords, N_coords * 3 * sizeof(double));
    hipMalloc((void**)&device_dists, N_grid_points * N_coords * sizeof(double));

    // Copy data to the device
    hipMemcpy(
        device_grid_pos, host_grid_pos, 
        N_grid_points * 3 * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_coords, host_coords, 
        N_coords * 3 * sizeof(double), 
        hipMemcpyHostToDevice
    );

    // Run the kernel
    dim3 dimBlock(32, 32);
    dim3 dimGrid(
        (N_grid_points + dimBlock.x - 1) / dimBlock.x,
        (N_coords + dimBlock.y - 1) / dimBlock.y
    );
    calc_pairwise_dist_kernel<<<dimGrid, dimBlock>>>(
        device_grid_pos, device_coords, N_coords, N_grid_points, device_dists
    );

    // Copy data back to the host
    hipMemcpy(
        host_dists, device_dists, 
        N_grid_points * N_coords * sizeof(double), 
        hipMemcpyDeviceToHost
    );

    // Free memory on the device
    hipFree(device_grid_pos);
    hipFree(device_coords);
    hipFree(device_dists);
}

extern "C"
void gen_elec_grid(
    double* host_dists, double* host_charges, const double cc_elec, 
    const double rad_dielec_const, const double elec_rep_max, 
    const double elec_attr_max, 
    const int N_coords, const int N_grid_points, 
    double* host_electrostat_grid
) {
    // Allocate memory on the device
    double* device_dists;
    double* device_charges;
    double* device_electrostat_grid;

    hipMalloc((void**)&device_dists, N_grid_points * N_coords * sizeof(double));
    hipMalloc((void**)&device_charges, N_coords * sizeof(double));
    hipMalloc((void**)&device_electrostat_grid, N_grid_points * sizeof(double));

    // Copy data to the device
    hipMemcpy(
        device_dists, host_dists, 
        N_grid_points * N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_charges, host_charges, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );

    // Run the kernel
    // dim3 dimBlock(32, 32);
    // dim3 dimGrid(
    //     (N_grid_points + dimBlock.x - 1) / dimBlock.x
    //     // (N_coords + dimBlock.y - 1) / dimBlock.y
    // );
    int block_size = 32;
    int grid_size = (N_grid_points + block_size - 1) / block_size;
    gen_elec_grid_kernel<<<grid_size, block_size>>>(
        device_dists, device_charges, cc_elec, 
        rad_dielec_const, elec_rep_max, elec_attr_max, 
        N_coords, N_grid_points, device_electrostat_grid
    );

    // Copy data back to the host
    hipMemcpy(
        host_electrostat_grid, device_electrostat_grid, 
        N_grid_points * sizeof(double), 
        hipMemcpyDeviceToHost
    );

    // Free memory on the device
    hipFree(device_dists);
    hipFree(device_charges);
    hipFree(device_electrostat_grid);
}

extern "C"
void gen_vdw_grid(
    double* host_dists, double* host_epsilons, double* host_vdw_rs, 
    const double probe_radius, const double vwd_softcore_max, 
    const int N_coords, const int N_grid_points, double* host_vdw_grid
) {
    // Allocate memory on the device
    double* device_dists;
    double* device_epsilons;
    double* device_vdw_rs;
    double* device_vdw_grid;

    hipMalloc((void**)&device_dists, N_grid_points * N_coords * sizeof(double));
    hipMalloc((void**)&device_epsilons, N_coords * sizeof(double));
    hipMalloc((void**)&device_vdw_rs, N_coords * sizeof(double));
    hipMalloc((void**)&device_vdw_grid, N_grid_points * sizeof(double));

    // Copy data to the device
    hipMemcpy(
        device_dists, host_dists, 
        N_grid_points * N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_epsilons, host_epsilons, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_vdw_rs, host_vdw_rs, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );

    // Run the kernel
    // dim3 dimBlock(32, 32);
    // dim3 dimGrid(
    //     (N_grid_points + dimBlock.x - 1) / dimBlock.x 
    //     // (N_coords + dimBlock.y - 1) / dimBlock.y
    // );
    int block_size = 32;
    int grid_size = (N_grid_points + block_size - 1) / block_size;
    gen_vdw_grid_kernel<<<grid_size, block_size>>>(
        device_dists, device_epsilons, device_vdw_rs, 
        probe_radius, vwd_softcore_max, 
        N_coords, N_grid_points, device_vdw_grid
    );

    // Copy data back to the host
    hipMemcpy(
        host_vdw_grid, device_vdw_grid, 
        N_grid_points * sizeof(double), 
        hipMemcpyDeviceToHost
    );

    // Free memory on the device
    hipFree(device_dists);
    hipFree(device_epsilons);
    hipFree(device_vdw_rs);
    hipFree(device_vdw_grid);
}

extern "C"
void gen_all_grids(
    const double* host_grid_pos, const double* host_coords, const double* host_charges, 
    const double* host_epsilons, const double* host_vdw_rs, const double cc_elec, 
    const double rad_dielec_const, const double elec_rep_max, 
    const double elec_attr_max, const double probe_radius,
    const double vwd_softcore_max, const int N_coords, const int N_grid_points,
    double* host_dists, double* host_electrostat_grid, double* host_vdw_grid
){
    // Allocate memory on the device
    // input
    double* device_grid_pos;
    double* device_coords;
    double* device_charges;
    double* device_epsilons;
    double* device_vdw_rs;
    // output
    double* device_dists;
    double* device_electrostat_grid;
    double* device_vdw_grid;

    hipMalloc((void**)&device_grid_pos, N_grid_points * 3 * sizeof(double));
    hipMalloc((void**)&device_coords, N_coords * 3 * sizeof(double));
    hipMalloc((void**)&device_epsilons, N_coords * sizeof(double));
    hipMalloc((void**)&device_vdw_rs, N_coords * sizeof(double));
    hipMalloc((void**)&device_vdw_grid, N_grid_points * sizeof(double));
    hipMalloc((void**)&device_dists, N_grid_points * N_coords * sizeof(double));
    hipMalloc((void**)&device_charges, N_coords * sizeof(double));
    hipMalloc((void**)&device_electrostat_grid, N_grid_points * sizeof(double));

    // Copy data to the device
    hipMemcpy(
        device_grid_pos, host_grid_pos, 
        N_grid_points * 3 * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_coords, host_coords, 
        N_coords * 3 * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_charges, host_charges, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_epsilons, host_epsilons, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_vdw_rs, host_vdw_rs, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );

    // Run the kernel
    dim3 dimBlock(32, 32);
    dim3 dimGrid(
        (N_grid_points + dimBlock.x - 1) / dimBlock.x,
        (N_coords + dimBlock.y - 1) / dimBlock.y
    );
    calc_pairwise_dist_kernel<<<dimGrid, dimBlock>>>(
        device_grid_pos, device_coords, N_coords, N_grid_points, device_dists
    );

    int block_size = 32;
    int grid_size = (N_grid_points + block_size - 1) / block_size;
    gen_elec_grid_kernel<<<grid_size, block_size>>>(
        device_dists, device_charges, cc_elec, 
        rad_dielec_const, elec_rep_max, elec_attr_max, 
        N_coords, N_grid_points, device_electrostat_grid
    );

    gen_vdw_grid_kernel<<<grid_size, block_size>>>(
        device_dists, device_epsilons, device_vdw_rs, 
        probe_radius, vwd_softcore_max, 
        N_coords, N_grid_points, device_vdw_grid
    );

    // Copy data back to the host
    hipMemcpy(
        host_dists, device_dists, 
        N_grid_points * N_coords * sizeof(double), 
        hipMemcpyDeviceToHost
    );
    hipMemcpy(
        host_electrostat_grid, device_electrostat_grid, 
        N_grid_points * sizeof(double), 
        hipMemcpyDeviceToHost
    );
    hipMemcpy(
        host_vdw_grid, device_vdw_grid, 
        N_grid_points * sizeof(double), 
        hipMemcpyDeviceToHost
    );

    // Free memory on the device
    hipFree(device_grid_pos);
    hipFree(device_coords);
    hipFree(device_charges);
    hipFree(device_epsilons);
    hipFree(device_vdw_rs);
    hipFree(device_dists);
    hipFree(device_electrostat_grid);
    hipFree(device_vdw_grid);
}