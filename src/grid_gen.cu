// This file contains the CUDA kernels for calculating pairwise distances,
// electrostatic grids and vdw grids. This version is for CUDA device of 
// compute capability above sm_6X (pascal), where atomicAdd supports 
// double precision.
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <hip/hip_runtime.h>


// Helper function for checking CUDA errors
#define CUDA_CHECK(call) \
do { \
    hipError_t result = call; \
    if (result != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", \
        __FILE__, __LINE__, result, hipGetErrorString(result), #call); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

__global__ void calc_pairwise_dist_kernel(
    double* grid_pos, double* coords, int N_coords, int N_grid_points, 
    double* dists
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
  
    if (i < N_grid_points && j < N_coords) {
        double dx, dy, dz;

        dx = grid_pos[i * 3] - coords[j * 3];
        dy = grid_pos[i * 3 + 1] - coords[j * 3 + 1];
        dz = grid_pos[i * 3 + 2] - coords[j * 3 + 2];

        dists[i * N_coords + j] = sqrtf(dx * dx + dy * dy + dz * dz);
    }
}

__device__ double calc_point_elec_potential(
    double dist, double elec_const, double charge, double rc, double alpha,
    double elec_rep_max, double elec_attr_max
) {
    double cur_potential;
    
    if (dist > rc) {
        cur_potential = elec_const / (dist * dist);
    } else {
        double alpha_tmp = alpha * dist * dist;
        if (charge > 0) {
            cur_potential = elec_rep_max - alpha_tmp;
        } else {
            cur_potential = elec_attr_max + alpha_tmp;
        }
    }
    
    return cur_potential;
}

__global__ void gen_elec_grid_kernel(
    const double* dists, const double* charges, const double cc_elec,
    const double rad_dielec_const, const double elec_rep_max, 
    const double elec_attr_max,
    const int N_coords, const int N_grid_points, double* electrostat_grid
) { 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N_grid_points && j < N_coords) {
        double elec_const, rc, alpha;
        double emax_tmp;
        double cur_grid_val = 0.0;
        double dist = dists[i * N_coords + j];

        elec_const = cc_elec * charges[j] / rad_dielec_const;

        if (charges[j] > 0) {
            emax_tmp = elec_rep_max;
        } else {
            emax_tmp = elec_attr_max;
        }
        rc = sqrt(2.0 * fabs(elec_const / emax_tmp));
        alpha = fabs(emax_tmp / (2.0 * rc * rc));

        double cur_potential = calc_point_elec_potential(
            dist, elec_const, charges[j], rc, alpha, 
            elec_rep_max, elec_attr_max
        );
        cur_grid_val += cur_potential;
        // for CUDA 8 and sm_6X above, atomicAdd supports double precision
        atomicAdd(&electrostat_grid[i], cur_grid_val);
    }
}

__device__ double calc_point_vdw_potential(
    double dist, double eps_sqrt, double r_min, double probe_radius, 
    double vwd_softcore_max, double rc_vdw, double beta
) {
    double cur_potential;

    double r_min_over_dist = r_min / dist;
    if (dist > rc_vdw) {
        cur_potential = (
            eps_sqrt * (
                powf(r_min_over_dist, 12.0) - 2.0 * powf(r_min_over_dist, 6.0)
            )
        );
    } else {
        cur_potential = (
            vwd_softcore_max * (1.0 - 0.5 * powf((dist / rc_vdw), beta))
        );
    }

    return cur_potential;
}

__global__ void gen_vdw_grid_kernel(
    double* dists, double* epsilons, double* vdw_rs, double probe_radius, 
    double vwd_softcore_max, int N_coords, int N_grid_points, double* vdw_grid
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < N_grid_points && j < N_coords) {
        double r_min, eps_sqrt, vdwconst, rc_vdw, beta;
        double dist = dists[i * N_coords + j];

        r_min = vdw_rs[j] + probe_radius;
        eps_sqrt = sqrt(fabs(epsilons[j]));
        vdwconst = 1.0 + sqrt(1.0 + 0.5 * fabs(vwd_softcore_max) / eps_sqrt);
        rc_vdw = r_min * powf(vdwconst, -1.0 / 6.0);
        beta = 24.0 * eps_sqrt / 
        vwd_softcore_max * (vdwconst * vdwconst - vdwconst);

        double cur_potential = calc_point_vdw_potential(
            dist, eps_sqrt, r_min, probe_radius, 
            vwd_softcore_max, rc_vdw, beta
        );
        // for CUDA 8 and sm_6X above, atomicAdd supports double precision
        atomicAdd(&vdw_grid[i], cur_potential);
    }
}

void calc_chunk_pairwise_dist(
    double* host_grid_pos, double* host_coords, 
    const int N_coords, const int N_grid_points, size_t chunk_size,
    size_t num_chunks, double* host_dists
) {
    double* device_grid_pos;
    double* device_coords;
    double* device_dists;

    hipMalloc((void**)&device_coords, N_coords * 3 * sizeof(double));
    hipMemcpy(
        device_coords, host_coords,
        N_coords * 3 * sizeof(double), 
        hipMemcpyHostToDevice
    );

    hipMalloc((void**)&device_grid_pos, chunk_size * 3 * sizeof(double));
    CUDA_CHECK(hipPeekAtLastError());
    hipMalloc((void**)&device_dists, chunk_size * N_coords * sizeof(double));
    CUDA_CHECK(hipPeekAtLastError());
    
    for (size_t i = 0; i < num_chunks; ++i) {
        size_t start = i * chunk_size;
        size_t end = std::min<size_t>((i + 1) * chunk_size, N_grid_points);
        size_t cur_chunk_size = end - start;
        printf("Chunk %zu num of grid points: %zu\n", i, cur_chunk_size);
        // Copy chunk to device
        hipMemcpy(
            device_grid_pos, &host_grid_pos[start], 
            cur_chunk_size * sizeof(double), 
            hipMemcpyHostToDevice
        );
        CUDA_CHECK(hipPeekAtLastError());
        // cudaMemset(device_dists, 0, chunk_size * N_coords * sizeof(double));
        // Run the kernel
        dim3 dimBlock(32, 32);
        dim3 dimGrid(
            (cur_chunk_size + dimBlock.x - 1) / dimBlock.x,
            (N_coords + dimBlock.y - 1) / dimBlock.y
        );
        calc_pairwise_dist_kernel<<<dimGrid, dimBlock>>>(
            device_grid_pos, device_coords, 
            N_coords, cur_chunk_size, device_dists
        );
        CUDA_CHECK(hipPeekAtLastError());
        hipMemcpy(
            &host_dists[start * N_coords], device_dists,
            cur_chunk_size * N_coords * sizeof(double),
            hipMemcpyDeviceToHost
        );
        CUDA_CHECK(hipPeekAtLastError());
    }
    hipFree(device_dists);
    hipFree(device_grid_pos);
    hipFree(device_coords);
}

extern "C"
void calc_all_pairwise_dist(
    double* host_grid_pos, double* host_coords, 
    const int N_coords, const int N_grid_points, double* host_dists
) {
    // Allocate memory on the device
    double* device_grid_pos;
    double* device_coords;
    double* device_dists;

    hipMalloc((void**)&device_grid_pos, N_grid_points * 3 * sizeof(double));
    hipMalloc((void**)&device_coords, N_coords * 3 * sizeof(double));
    hipMalloc((void**)&device_dists, N_grid_points * N_coords * sizeof(double));

    // Copy data to the device
    hipMemcpy(
        device_grid_pos, host_grid_pos, 
        N_grid_points * 3 * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_coords, host_coords, 
        N_coords * 3 * sizeof(double), 
        hipMemcpyHostToDevice
    );

    // Run the kernel
    dim3 dimBlock(32, 32);
    dim3 dimGrid(
        (N_grid_points + dimBlock.x - 1) / dimBlock.x,
        (N_coords + dimBlock.y - 1) / dimBlock.y
    );
    calc_pairwise_dist_kernel<<<dimGrid, dimBlock>>>(
        device_grid_pos, device_coords, N_coords, N_grid_points, device_dists
    );

    // Copy data back to the host
    hipMemcpy(
        host_dists, device_dists, 
        N_grid_points * N_coords * sizeof(double), 
        hipMemcpyDeviceToHost
    );

    // Free memory on the device
    hipFree(device_grid_pos);
    hipFree(device_coords);
    hipFree(device_dists);
}

extern "C"
void calc_pairwise_dist(
    double* host_grid_pos, double* host_coords, 
    const int N_coords, const int N_grid_points, double* host_dists
){
    size_t dists_size = N_grid_points * N_coords * sizeof(double);
    size_t coords_size = N_coords * 3 * sizeof(double);
    size_t grid_pos_size = N_grid_points * 3 * sizeof(double);

    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    size_t device_memory = prop.totalGlobalMem;
    size_t leftover_memory = device_memory - coords_size - grid_pos_size;

    if (dists_size > leftover_memory){
        // 1.1 is a safety factor for 10% headroom
        size_t chunk_size = leftover_memory / (N_coords * 1.1 * sizeof(double));
        size_t num_chunks = (N_grid_points + chunk_size - 1) / chunk_size;
        printf(
            "Array size is too large for the device memory (%zuMB). Split into %zu chunks\n", 
            leftover_memory/1024/1024, num_chunks
        );
        calc_chunk_pairwise_dist(
            host_grid_pos, host_coords, 
            N_coords, N_grid_points, chunk_size, num_chunks, host_dists
        );
    } else {
        calc_all_pairwise_dist(
            host_grid_pos, host_coords, 
            N_coords, N_grid_points, host_dists);
    }
}

extern "C"
void gen_elec_grid(
    double* host_dists, double* host_charges, const double cc_elec, 
    const double rad_dielec_const, const double elec_rep_max, 
    const double elec_attr_max, 
    const int N_coords, const int N_grid_points, 
    double* host_electrostat_grid
) {
    // Allocate memory on the device
    double* device_dists;
    double* device_charges;
    double* device_electrostat_grid;

    hipMalloc((void**)&device_dists, N_grid_points * N_coords * sizeof(double));
    hipMalloc((void**)&device_charges, N_coords * sizeof(double));
    hipMalloc((void**)&device_electrostat_grid, N_grid_points * sizeof(double));

    // Copy data to the device
    hipMemcpy(
        device_dists, host_dists, 
        N_grid_points * N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_charges, host_charges, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );

    // Run the kernel
    dim3 dimBlock(32, 32);
    dim3 dimGrid(
        (N_grid_points + dimBlock.x - 1) / dimBlock.x,
        (N_coords + dimBlock.y - 1) / dimBlock.y
    );
    gen_elec_grid_kernel<<<dimGrid, dimBlock>>>(
        device_dists, device_charges, cc_elec, 
        rad_dielec_const, elec_rep_max, elec_attr_max, 
        N_coords, N_grid_points, device_electrostat_grid
    );

    // Copy data back to the host
    hipMemcpy(
        host_electrostat_grid, device_electrostat_grid, 
        N_grid_points * sizeof(double), 
        hipMemcpyDeviceToHost
    );

    // Free memory on the device
    hipFree(device_dists);
    hipFree(device_charges);
    hipFree(device_electrostat_grid);
}

extern "C"
void gen_vdw_grid(
    double* host_dists, double* host_epsilons, double* host_vdw_rs, 
    const double probe_radius, const double vwd_softcore_max, 
    const int N_coords, const int N_grid_points, double* host_vdw_grid
) {
    // Allocate memory on the device
    double* device_dists;
    double* device_epsilons;
    double* device_vdw_rs;
    double* device_vdw_grid;

    hipMalloc((void**)&device_dists, N_grid_points * N_coords * sizeof(double));
    hipMalloc((void**)&device_epsilons, N_coords * sizeof(double));
    hipMalloc((void**)&device_vdw_rs, N_coords * sizeof(double));
    hipMalloc((void**)&device_vdw_grid, N_grid_points * sizeof(double));

    // Copy data to the device
    hipMemcpy(
        device_dists, host_dists, 
        N_grid_points * N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_epsilons, host_epsilons, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_vdw_rs, host_vdw_rs, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );

    // Run the kernel
    dim3 dimBlock(32, 32);
    dim3 dimGrid(
        (N_grid_points + dimBlock.x - 1) / dimBlock.x, 
        (N_coords + dimBlock.y - 1) / dimBlock.y
    );
    gen_vdw_grid_kernel<<<dimGrid, dimBlock>>>(
        device_dists, device_epsilons, device_vdw_rs, 
        probe_radius, vwd_softcore_max, 
        N_coords, N_grid_points, device_vdw_grid
    );

    // Copy data back to the host
    hipMemcpy(
        host_vdw_grid, device_vdw_grid, 
        N_grid_points * sizeof(double), 
        hipMemcpyDeviceToHost
    );

    // Free memory on the device
    hipFree(device_dists);
    hipFree(device_epsilons);
    hipFree(device_vdw_rs);
    hipFree(device_vdw_grid);
}

extern "C"
void gen_all_grids(
    const double* host_grid_pos, const double* host_coords, const double* host_charges, 
    const double* host_epsilons, const double* host_vdw_rs, const double cc_elec, 
    const double rad_dielec_const, const double elec_rep_max, 
    const double elec_attr_max, const double probe_radius,
    const double vwd_softcore_max, const int N_coords, const int N_grid_points,
    double* host_dists, double* host_electrostat_grid, double* host_vdw_grid
){
    // Allocate memory on the device
    // input
    double* device_grid_pos;
    double* device_coords;
    double* device_charges;
    double* device_epsilons;
    double* device_vdw_rs;
    // output
    double* device_dists;
    double* device_electrostat_grid;
    double* device_vdw_grid;

    hipMalloc((void**)&device_grid_pos, N_grid_points * 3 * sizeof(double));
    hipMalloc((void**)&device_coords, N_coords * 3 * sizeof(double));
    hipMalloc((void**)&device_charges, N_coords * sizeof(double));
    hipMalloc((void**)&device_epsilons, N_coords * sizeof(double));
    hipMalloc((void**)&device_vdw_rs, N_coords * sizeof(double));

    hipMalloc((void**)&device_dists, N_grid_points * N_coords * sizeof(double));
    hipMalloc((void**)&device_electrostat_grid, N_grid_points * sizeof(double));
    hipMalloc((void**)&device_vdw_grid, N_grid_points * sizeof(double));


    // Copy data to the device
    hipMemcpy(
        device_grid_pos, host_grid_pos, 
        N_grid_points * 3 * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_coords, host_coords, 
        N_coords * 3 * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_charges, host_charges, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_epsilons, host_epsilons, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );
    hipMemcpy(
        device_vdw_rs, host_vdw_rs, 
        N_coords * sizeof(double), 
        hipMemcpyHostToDevice
    );

    // Run the kernel
    dim3 dimBlock(32, 32);
    dim3 dimGrid(
        (N_grid_points + dimBlock.x - 1) / dimBlock.x,
        (N_coords + dimBlock.y - 1) / dimBlock.y
    );
    calc_pairwise_dist_kernel<<<dimGrid, dimBlock>>>(
        device_grid_pos, device_coords, N_coords, N_grid_points, device_dists
    );

    gen_elec_grid_kernel<<<dimGrid, dimBlock>>>(
        device_dists, device_charges, cc_elec, 
        rad_dielec_const, elec_rep_max, elec_attr_max, 
        N_coords, N_grid_points, device_electrostat_grid
    );

    gen_vdw_grid_kernel<<<dimGrid, dimBlock>>>(
        device_dists, device_epsilons, device_vdw_rs, 
        probe_radius, vwd_softcore_max, 
        N_coords, N_grid_points, device_vdw_grid
    );

    // Copy data back to the host
    hipMemcpy(
        host_dists, device_dists, 
        N_grid_points * N_coords * sizeof(double), 
        hipMemcpyDeviceToHost
    );
    hipMemcpy(
        host_electrostat_grid, device_electrostat_grid, 
        N_grid_points * sizeof(double), 
        hipMemcpyDeviceToHost
    );
    hipMemcpy(
        host_vdw_grid, device_vdw_grid, 
        N_grid_points * sizeof(double), 
        hipMemcpyDeviceToHost
    );

    // Free memory on the device
    hipFree(device_grid_pos);
    hipFree(device_coords);
    hipFree(device_charges);
    hipFree(device_epsilons);
    hipFree(device_vdw_rs);
    hipFree(device_dists);
    hipFree(device_electrostat_grid);
    hipFree(device_vdw_grid);
}